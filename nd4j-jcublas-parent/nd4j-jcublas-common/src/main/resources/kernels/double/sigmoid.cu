extern "C"



#include <hip/hip_runtime.h>
#include <math.h>


__global__ void sigmoid_double(int n,int idx,double *dy,int incy,double *result) {
        for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
                       if(i >= idx && i % incy == 0)
                           result[i] =  1 / (1 + exp(-dy[i]));
         }

 }